#include <hip/hip_runtime.h>

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

extern "C" void thrust_reduce_wrapper(
	double * __restrict__ c,
	int n,
	double * sum
) {
	thrust::device_ptr<double> c_ptr = thrust::device_pointer_cast(c);
	*sum = thrust::reduce(c_ptr, c_ptr + n);
}
