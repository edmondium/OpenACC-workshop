#include <hip/hip_runtime.h>

#include <thrust/device_ptr.h>
#include <thrust/reduce.h>

extern "C" void thrust_reduce_wrapper(
	double * __restrict__ c,
	int n,
	double * sum
) {
	thrust::device_ptr<double> c_ptr = thrust::device_pointer_cast(c);
	// TODO: Call thrust::reduce using c_ptr (the length is n) and give thrust's output to the calling function by means of sum (which needs to be dereferenced)
	// Hint: thrust::reduce can use device_ptr address as InputIterators
	// Documentation: https://thrust.github.io/doc/group__reductions.html#ga69434d74f2e6117040fb38d1a28016c2
}
